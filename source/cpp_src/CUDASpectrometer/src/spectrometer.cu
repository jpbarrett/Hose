#include "hip/hip_runtime.h"
/* Project that takes signal and calculates averaged power spectrum using NVIDIA
   CUDA GPGPU programming. 

   Juha Vierinen (x@mit.edu)
   Cory Cotter (optimization of square and accumulate sum)
   John Barrett (added support for signed int, stripped down code to minimal library)
*/


#include "spectrometer.h"
#define MIN(X,Y) (((X) < (Y)) ? (X) : (Y))

void cuda_alloc_pinned_memory( void** ptr, size_t s)
{
    int code = hipHostMalloc(ptr, s);
    if(code != hipSuccess)
    {
        //set pointer to null if there is an error
        fprintf(stderr, "Cuda error: hipHostMalloc failed.\n");
        *ptr = NULL;
    }
}

void cuda_free_pinned_memory(void* ptr)
{
    int code = hipHostFree(ptr);
    if(code != hipSuccess)
    {
        fprintf(stderr, "Cuda error: hipHostFree failed.\n");
    }
}

extern "C" spectrometer_data *new_spectrometer_data(int data_length, int spectrum_length)
{
  spectrometer_data *d;
  int n_spectra;
  n_spectra=data_length/spectrum_length;

  d=(spectrometer_data *)malloc(sizeof(spectrometer_data));

  // result on the cpu
  d->spectrum = (float *)malloc( (spectrum_length/2+1) * sizeof(float));

  d->data_length=data_length;
  d->spectrum_length=spectrum_length;
  d->n_spectra=n_spectra;

  d->window = (float *)malloc(spectrum_length*sizeof(float));
  blackmann_harris(d->window,spectrum_length);

  // allocating device memory to the above pointers
  // reserve extra for +1 in place transforms
  if (hipMalloc((void **) &d->d_in, sizeof(hipfftComplex)*n_spectra*(spectrum_length/2 + 1)) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate input data vector\n");
    exit(EXIT_FAILURE);
  }
  if (hipMalloc((void **) &d->d_z_out, sizeof(hipfftComplex)*n_spectra*(spectrum_length/2 + 1)) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate input data vector\n");
    exit(EXIT_FAILURE);
  }

  if (hipMalloc((void **) &d->d_window, sizeof(float)*spectrum_length) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate input data vector\n");
    exit(EXIT_FAILURE);
  }

  // in-place seems to have a bug that causes the end to be garbled.
  //  d->d_z_out =(hipfftComplex *) d->d_in;

  if (hipMalloc((void **) &d->ds_in,sizeof(uint16_t)*data_length) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate input data vector\n");
    exit(EXIT_FAILURE);
  }

  if (hipMalloc((void **) &d->d_spectrum,sizeof(float)*(spectrum_length/2+1))
      != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate spectrum\n");
    exit(EXIT_FAILURE);
  }

  // initializing 1D FFT plan, this will tell cufft execution how to operate
  // cufft is well optimized and will run with different parameters than above
  //    hipfftHandle plan;
  if (hipfftPlan1d(&d->plan, spectrum_length, HIPFFT_R2C, n_spectra) != HIPFFT_SUCCESS) 
  {
    fprintf(stderr, "CUFFT error: Plan creation failed\n");
    exit(EXIT_FAILURE);
  }

  // copy window to device
  if (hipMemcpy(d->d_window, d->window, sizeof(float)*spectrum_length, hipMemcpyHostToDevice) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Memory copy failed, window function HtD\n");
    exit(EXIT_FAILURE);
  }
  return(d);
}

extern "C" void free_spectrometer_data(spectrometer_data *d)
{
  if (hipFree(d->d_in) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free input\n");
    exit(EXIT_FAILURE);
  }
  free(d->window);
  free(d->spectrum);
  if (hipFree(d->d_window) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free window function\n");
    exit(EXIT_FAILURE);
  }
  if (hipFree(d->d_z_out) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free input\n");
    exit(EXIT_FAILURE);
  }
  if (hipFree(d->ds_in) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free input\n");
    exit(EXIT_FAILURE);
  }

  if (hipFree(d->d_spectrum) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free spec\n");
    exit(EXIT_FAILURE);
  }
  if (hipfftDestroy(d->plan) != HIPFFT_SUCCESS) {
    fprintf(stderr, "CUFFT error: Failed to destroy plan\n");
    exit(EXIT_FAILURE);
  }
  free(d);
}


extern "C" spectrometer_data_s *new_spectrometer_data_s(int data_length, int spectrum_length)
{
  spectrometer_data_s *d;
  int n_spectra;
  n_spectra=data_length/spectrum_length;

  unsigned long available, total;
  hipMemGetInfo(&available, &total);
    printf("gpu mem: avail and total = %lu and %lu \n", available, total);


  d=(spectrometer_data_s *)malloc(sizeof(spectrometer_data_s));

  // result on the cpu
  d->spectrum = (float *)malloc( (spectrum_length/2+1) * sizeof(float));

  d->data_length=data_length;
  d->spectrum_length=spectrum_length;
  d->n_spectra=n_spectra;

  d->window = (float *)malloc(spectrum_length*sizeof(float));
  blackmann_harris(d->window,spectrum_length);

  // allocating device memory to the above pointers
  // reserve extra for +1 in place transforms
    unsigned long want =  sizeof(hipfftComplex)*n_spectra*(spectrum_length/2 + 1);
    printf("required gpu buff size = %lu \n", want);


  int code = hipMalloc( (void **) &d->d_in, sizeof(hipfftComplex)*n_spectra*(spectrum_length/2 + 1) );
  if( code != hipSuccess)
  {
    printf("code = %d\n", code);
    fprintf(stderr, "Cuda error: Failed to allocate input data vector\n");
    exit(EXIT_FAILURE);
  }

  hipMemGetInfo(&available, &total);
    printf("gpu mem: avail and total = %lu and %lu\n", available, total);


  code = hipMalloc((void **) &d->d_z_out, sizeof(hipfftComplex)*n_spectra*(spectrum_length/2 + 1));
  if ( code != hipSuccess)
  {
    printf("code = %d\n", code);
    fprintf(stderr, "Cuda error: Failed to allocate input data vector\n");
    exit(EXIT_FAILURE);
  }

  hipMemGetInfo(&available, &total);
    printf("gpu mem: avail and total = %lu and %lu\n", available, total);


    code = hipMalloc((void **) &d->d_window, sizeof(float)*spectrum_length);
  if (code != hipSuccess)
  {
    printf("code = %d\n", code);
    fprintf(stderr, "Cuda error: Failed to allocate input data vector\n");
    exit(EXIT_FAILURE);
  }

  hipMemGetInfo(&available, &total);
    printf("gpu mem: avail and total = %lu and %lu\n", available, total);


  // in-place seems to have a bug that causes the end to be garbled.
  //  d->d_z_out =(hipfftComplex *) d->d_in;

    code = hipMalloc((void **) &d->ds_in,sizeof(int16_t)*data_length);
  if (code != hipSuccess)
  {
    printf("code = %d\n", code);
    fprintf(stderr, "Cuda error: Failed to allocate input data vector\n");
    exit(EXIT_FAILURE);
  }

  hipMemGetInfo(&available, &total);
    printf("gpu mem: avail and total = %lu and %lu\n", available, total);


  if (hipMalloc((void **) &d->d_spectrum,sizeof(float)*(spectrum_length/2+1))
      != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate spectrum\n");
    exit(EXIT_FAILURE);
  }

  hipMemGetInfo(&available, &total);
    printf("gpu mem: avail and total = %lu and %lu\n", available, total);


  // initializing 1D FFT plan, this will tell cufft execution how to operate
  // cufft is well optimized and will run with different parameters than above
  //    hipfftHandle plan;
    
  if (hipfftPlan1d(&d->plan, spectrum_length, HIPFFT_R2C, n_spectra) != HIPFFT_SUCCESS)
  {
    fprintf(stderr, "CUFFT error: Plan creation failed\n");
    fprintf(stderr, "spec len, n spec = %d, %d", spectrum_length, n_spectra);
    exit(EXIT_FAILURE);
  }

  // copy window to device
  if (hipMemcpy(d->d_window, d->window, sizeof(float)*spectrum_length, hipMemcpyHostToDevice) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Memory copy failed, window function HtD\n");
    exit(EXIT_FAILURE);
  }
  return(d);
}

extern "C" void free_spectrometer_data_s(spectrometer_data_s *d)
{
  if (hipFree(d->d_in) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free input\n");
    exit(EXIT_FAILURE);
  }
  free(d->window);
  free(d->spectrum);
  if (hipFree(d->d_window) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free window function\n");
    exit(EXIT_FAILURE);
  }
  if (hipFree(d->d_z_out) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free input\n");
    exit(EXIT_FAILURE);
  }
  if (hipFree(d->ds_in) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free input\n");
    exit(EXIT_FAILURE);
  }

  if (hipFree(d->d_spectrum) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free spec\n");
    exit(EXIT_FAILURE);
  }
  if (hipfftDestroy(d->plan) != HIPFFT_SUCCESS) {
    fprintf(stderr, "CUFFT error: Failed to destroy plan\n");
    exit(EXIT_FAILURE);
  }
  free(d);
}


/*
  create Blackmann-Harris window function
 */
void blackmann_harris(float* pOut, unsigned int num)
{
  const float a0      = 0.35875f;
  const float a1      = 0.48829f;
  const float a2      = 0.14128f;
  const float a3      = 0.01168f;

  unsigned int idx    = 0;
  while( idx < num )
  {
    pOut[idx]   = a0 - (a1 * cosf( (2.0f * M_PI * idx) / (num - 1) )) + (a2 * cosf( (4.0f * M_PI * idx) / (num - 1) )) - (a3 * cosf( (6.0f * M_PI * idx) / (num - 1) ));
    idx++;
  }
}

/*
  average spectra
 */
__global__ void square_and_accumulate_sum(const hipfftComplex* d_in,
					  float* d_out,
					  const unsigned n_spectra,
					  const unsigned spectrum_length)
{
    unsigned idx = threadIdx.x + blockDim.x*blockIdx.x;
    while (idx < spectrum_length)
    {
        float result = 0.0;
        for (int i = 0; i < n_spectra; i++)
        {
            int d_idx = i * spectrum_length + idx;
            result += d_in[d_idx].x * d_in[d_idx].x + d_in[d_idx].y * d_in[d_idx].y;
        }
        d_out[idx] = result;
        idx += gridDim.x * blockDim.x;
    }
}

/*
  convert int16_t data vector into single precision floating point (original range is -32768 to 32767)
  also apply window function *w
 */
__global__ void short_to_float_s(int16_t *ds, float *df, float *w, int n_spectra, int spectrum_length)
{
  for(int spec_idx=blockIdx.x; spec_idx < n_spectra ; spec_idx+=N_BLOCKS_S)
  {
    for(int freq_idx=threadIdx.x; freq_idx < spectrum_length ; freq_idx+=N_THREADS_S)
    {
      //map signed shorts into range [-0.5,0.5)
      int idx=spec_idx*spectrum_length + freq_idx;
      df[idx] = w[freq_idx]*( (float)ds[idx] ) / 65535.0;
    }
  }
}

/*
  convert uint16_t data vector into single precision floating point (original range is 0 to 65535)
  also apply window function *w
 */
__global__ void short_to_float(uint16_t *ds, float *df, float *w, int n_spectra, int spectrum_length)
{
  for(int spec_idx=blockIdx.x; spec_idx < n_spectra ; spec_idx+=N_BLOCKS)
  {
    for(int freq_idx=threadIdx.x; freq_idx < spectrum_length ; freq_idx+=N_THREADS)
    {
      int idx=spec_idx*spectrum_length + freq_idx;
      //map unsigned shorts in to range [-0.5,0.5)
      df[idx] = w[freq_idx]*( (float)ds[idx] - 32768.0) / 65535.0;
    }
  }
}

/*
 transform and average signed data
*/

extern "C" void process_vector_no_output_s(int16_t *d_in, spectrometer_data_s *d)
{
    int n_spectra, data_length, spectrum_length;

    n_spectra=d->n_spectra;
    data_length=d->data_length;
    spectrum_length=d->spectrum_length;


    // ensure empty device spectrum
    if (hipMemsetAsync(d->d_spectrum, 0, sizeof(float)*(spectrum_length/2 + 1)) != hipSuccess)
    {
      fprintf(stderr, "Cuda error: Failed to zero device spectrum\n");
        exit(EXIT_FAILURE);
    }

    // copy mem to device
    if (hipMemcpyAsync(d->ds_in, d_in, sizeof(int16_t)*data_length, hipMemcpyHostToDevice) != hipSuccess)
    {
      fprintf(stderr, "Cuda error: Memory copy failed\n");
      exit(EXIT_FAILURE);
    }

    // convert datatype using GPU
    short_to_float_s<<< N_BLOCKS_S, N_THREADS_S >>>(d->ds_in, d->d_in, d->d_window, n_spectra, spectrum_length);

    // cufft kernel execution
    if (hipfftExecR2C(d->plan, (float *)d->d_in, (hipfftComplex *)d->d_z_out)
	!= HIPFFT_SUCCESS)
    {
      fprintf(stderr, "CUFFT error: ExecC2C Forward failed\n");
      exit(EXIT_FAILURE);
    }

    // this needs to be faster:
    square_and_accumulate_sum<<< 1, N_THREADS_S >>>(d->d_z_out, d->d_spectrum, n_spectra, spectrum_length/2+1);
    if (hipGetLastError() != hipSuccess) {
       fprintf(stderr, "Cuda error: Kernel failure, square_and_accumulate_sum\n");
       exit(EXIT_FAILURE);
    }

    // copying device resultant spectrum to host, now able to be manipulated
    if (hipMemcpy(d->spectrum, d->d_spectrum, sizeof(float) * spectrum_length/2,
        hipMemcpyDeviceToHost) != hipSuccess)
    {
        fprintf(stderr, "Cuda error: Memory copy failed, spectrum DtH\n");
        exit(EXIT_FAILURE);
    }

}


/*
 transform and average unsigned data
*/

extern "C" void process_vector_no_output(uint16_t *d_in, spectrometer_data *d)
{
    int n_spectra, data_length, spectrum_length;

    n_spectra=d->n_spectra;
    data_length=d->data_length;
    spectrum_length=d->spectrum_length;


    // ensure empty device spectrum
    if (hipMemset(d->d_spectrum, 0, sizeof(float)*(spectrum_length/2 + 1)) != hipSuccess)
    {
      fprintf(stderr, "Cuda error: Failed to zero device spectrum\n");
        exit(EXIT_FAILURE);
    }

    // copy mem to device
    if (hipMemcpy(d->ds_in, d_in, sizeof(uint16_t)*data_length, hipMemcpyHostToDevice) != hipSuccess)
    {
      fprintf(stderr, "Cuda error: Memory copy failed: data len = %d\n", data_length);
      exit(EXIT_FAILURE);
    }

    // // convert datatype using GPU
    short_to_float<<< N_BLOCKS_S, N_THREADS_S >>>(d->ds_in, d->d_in, d->d_window, n_spectra, spectrum_length);

    // cufft kernel execution
    if (hipfftExecR2C(d->plan, (float *)d->d_in, (hipfftComplex *)d->d_z_out)
	!= HIPFFT_SUCCESS)
    {
      fprintf(stderr, "CUFFT error: ExecC2C Forward failed\n");
      exit(EXIT_FAILURE);
    }

    // this needs to be faster:
    square_and_accumulate_sum<<< 1, N_THREADS_S >>>(d->d_z_out, d->d_spectrum, n_spectra, spectrum_length/2+1);
    if (hipGetLastError() != hipSuccess) {
       fprintf(stderr, "Cuda error: Kernel failure, square_and_accumulate_sum\n");
       exit(EXIT_FAILURE);
    }

    // copying device resultant spectrum to host, now able to be manipulated
    if (hipMemcpy(d->spectrum, d->d_spectrum, sizeof(float) * spectrum_length/2,
        hipMemcpyDeviceToHost) != hipSuccess)
    {
        fprintf(stderr, "Cuda error: Memory copy failed, spectrum DtH\n");
        exit(EXIT_FAILURE);
    }

}
