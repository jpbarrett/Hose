#include "hip/hip_runtime.h"
/* Project that takes signal and calculates averaged power spectrum using NVIDIA
   CUDA GPGPU programming.

   Juha Vierinen (x@mit.edu)
   Cory Cotter (optimization of square and accumulate sum)
   John Barrett (added support for signed/unsigned int, stripped down code to minimal library) 2019
   John Barrett (added noise power calculation, re-org to clean up code) 2021
*/


#include "spectrometer.h"
#include "noise_statistics_mbp_reduce.h"


#define BOXCAR_WIN 0
#define BLACKMAN_HARRIS_WIN 1
#define HANN_WIN 2

/*
  create box-car (effectively no) window function
 */
void boxcar(float* pOut, unsigned int num)
{

  unsigned int idx    = 0;
  while( idx < num )
  {
    pOut[idx] = 1.0;
    idx++;
  }
}


/*
  create Blackmann-Harris window function
 */
void blackmann_harris(float* pOut, unsigned int num)
{
  const float a0      = 0.35875f;
  const float a1      = 0.48829f;
  const float a2      = 0.14128f;
  const float a3      = 0.01168f;

  unsigned int idx    = 0;
  while( idx < num )
  {
    pOut[idx]   = a0 - (a1 * cosf( (2.0f * M_PI * idx) / (num - 1) )) + (a2 * cosf( (4.0f * M_PI * idx) / (num - 1) )) - (a3 * cosf( (6.0f * M_PI * idx) / (num - 1) ));
    idx++;
  }
}


/*
  create a Hann window function
 */
void hann_window(float* pOut, unsigned int num)
{
  unsigned int idx    = 0;
  while( idx < num )
  {
    pOut[idx] = 0.5 + 0.5*( cosf( (2.0f * M_PI * idx) / (num - 1) ) );
    idx++;
  }
}

/*
create a Hamming window function
*/
void hamming_window(float* pOut, unsigned int num)
{
    unsigned int idx    = 0;
    while( idx < num )
    {
      pOut[idx] = 0.53836 + 0.46164*( cosf( (2.0f * M_PI * idx) / (num - 1) ) );
      idx++;
    }
}



void cuda_alloc_pinned_memory( void** ptr, size_t s)
{
    int code = hipHostMalloc(ptr, s);
    if(code != hipSuccess)
    {
        //set pointer to null if there is an error
        fprintf(stderr, "Cuda error: hipHostMalloc failed.\n");
        *ptr = NULL;
    }
}

void cuda_free_pinned_memory(void* ptr)
{
    int code = hipHostFree(ptr);
    if(code != hipSuccess)
    {
        fprintf(stderr, "Cuda error: hipHostFree failed.\n");
    }
}

void wrapped_cuda_malloc(void** ptr, size_t s)
{
    int code =hipMalloc(ptr, s);
    if( code != hipSuccess)
    {
        printf("code = %d\n", code);
        fprintf(stderr, "Cuda error: Failed to allocate input data vector\n");
        exit(EXIT_FAILURE);
    }
}

void print_cuda_meminfo()
{
    unsigned long available, total;
    hipMemGetInfo(&available, &total);
    printf("gpu mem: avail and total = %lu and %lu\n", available, total);
}

void wrapped_cuda_free(void* ptr)
{
    if (hipFree(ptr) != hipSuccess)
    {
        fprintf(stderr, "Cuda error: Failed to free input\n");
        exit(EXIT_FAILURE);
    }
}

extern "C" spectrometer_data *new_spectrometer_data(int data_length, int spectrum_length, int window_flag=BLACKMAN_HARRIS_WIN)
{
    spectrometer_data *d;
    int n_spectra;
    n_spectra = data_length/spectrum_length;
    print_cuda_meminfo();

    d = (spectrometer_data *) malloc(sizeof(spectrometer_data));

    // result on the cpu
    d->spectrum = (float*) malloc( (spectrum_length/2+1) * sizeof(float));
    d->data_length = data_length;
    d->spectrum_length = spectrum_length;
    d->n_spectra = n_spectra;
    d->window = (float*) malloc(spectrum_length*sizeof(float));

    if(window_flag == BOXCAR_WIN)
    {
        boxcar(d->window,spectrum_length);
    }

    if(window_flag == BLACKMAN_HARRIS_WIN)
    {
        blackmann_harris(d->window,spectrum_length);
    }
    
    if(window_flag == HANN_WIN)
    {
        hann_window(d->window,spectrum_length);
    }

    // allocating device memory to the above pointers
    // reserve extra for +1 in place transforms
    unsigned long want =  sizeof(hipfftComplex)*n_spectra*(spectrum_length/2 + 1);
    printf("required gpu buff size = %lu \n", want);

    //allocate space for FFT input
    wrapped_cuda_malloc( (void **) &d->d_in, sizeof(hipfftComplex)*n_spectra*(spectrum_length/2 + 1) );

    print_cuda_meminfo();
    //space for the noise power data accumulation
    wrapped_cuda_malloc( (void **) &d->d_out, sizeof(float)*N_THREADS );
    wrapped_cuda_malloc( (void **) &d->d_out2, sizeof(float)*N_THREADS );
    wrapped_cuda_malloc( (void **) &d->f_out, sizeof(float) );
    wrapped_cuda_malloc( (void **) &d->f_out2, sizeof(float) );
    print_cuda_meminfo();

    //allocate space for the x-formed spectra output
    // in-place seems to have a bug that causes the end to be garbled.
    //  d->d_z_out =(hipfftComplex *) d->d_in;
    wrapped_cuda_malloc( (void **) &d->d_z_out, sizeof(hipfftComplex)*n_spectra*(spectrum_length/2 + 1));
    print_cuda_meminfo();

    //allocate space for BH-window weights
    //note that we do not need multiple copies of the window weights
    //(we could replace this with pointer to a shared buffer)
    wrapped_cuda_malloc( (void **) &d->d_window, sizeof(float)*spectrum_length);
    print_cuda_meminfo();

    //allocate space to store the digitizer samples
    wrapped_cuda_malloc( (void **) &d->ds_in,sizeof(SAMPLE_TYPE)*data_length );
    print_cuda_meminfo();

    //allocate space for the power spectrum
    wrapped_cuda_malloc( (void **) &d->d_spectrum,sizeof(float)*(spectrum_length/2+1) );
    print_cuda_meminfo();

    // initializing 1D FFT plan, this will tell cufft execution how to operate
    // cufft is well optimized and will run with different parameters than above
    //    hipfftHandle plan;

    if (hipfftPlan1d(&d->plan, spectrum_length, HIPFFT_R2C, n_spectra) != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "CUFFT error: Plan creation failed\n");
        fprintf(stderr, "spec len, n spec = %d, %d", spectrum_length, n_spectra);
        exit(EXIT_FAILURE);
    }

    // copy window to device
    if (hipMemcpy(d->d_window, d->window, sizeof(float)*spectrum_length, hipMemcpyHostToDevice) != hipSuccess)
    {
        fprintf(stderr, "Cuda error: Memory copy failed, window function: host to device.\n");
        exit(EXIT_FAILURE);
    }
    return(d);
}

extern "C" void free_spectrometer_data(spectrometer_data *d)
{
    free(d->window);
    free(d->spectrum);
    wrapped_cuda_free(d->d_in);
    wrapped_cuda_free(d->d_window);
    wrapped_cuda_free(d->d_z_out);
    wrapped_cuda_free(d->ds_in);
    wrapped_cuda_free(d->d_out);
    wrapped_cuda_free(d->d_out2);
    wrapped_cuda_free(d->f_out);
    wrapped_cuda_free(d->f_out2);
    wrapped_cuda_free(d->d_spectrum);

    if (hipfftDestroy(d->plan) != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "CUFFT error: Failed to destroy plan\n");
        exit(EXIT_FAILURE);
    }

    free(d);
}


/*
  average spectra
 */
__global__ void square_and_accumulate_sum(const hipfftComplex* d_in,
					  float* d_out,
					  const unsigned n_spectra,
					  const unsigned spectrum_length)
{
    unsigned idx = threadIdx.x + blockDim.x*blockIdx.x;
    while (idx < spectrum_length)
    {
        float result = 0.0;
        for (int i = 0; i < n_spectra; i++)
        {
            int d_idx = i * spectrum_length + idx;
            result += d_in[d_idx].x * d_in[d_idx].x + d_in[d_idx].y * d_in[d_idx].y;
        }
        d_out[idx] = result;
        idx += gridDim.x * blockDim.x;
    }
}

/*
  convert int16_t data vector into single precision floating point (original range is -32768 to 32767)
  also apply window function *w
 */
__global__ void short_to_float_s(int16_t *ds, float *df, int n_spectra, int spectrum_length)
{
    for(int spec_idx=blockIdx.x; spec_idx < n_spectra ; spec_idx+=N_BLOCKS)
    {
        for(int freq_idx=threadIdx.x; freq_idx < spectrum_length ; freq_idx+=N_THREADS)
        {
            //map signed shorts into range [-0.5,0.5)
            //ADQ7DC range is 1Vpp (-0.5V to +0.5V)
            int idx=spec_idx*spectrum_length + freq_idx;
            df[idx] = ( (float)ds[idx] ) / 65535.0;
        }
    }
}

/*
  convert uint16_t data vector into single precision floating point (original range is 0 to 65535)
  also apply window function *w
 */
__global__ void short_to_float(uint16_t *ds, float *df, int n_spectra, int spectrum_length)
{
    for(int spec_idx=blockIdx.x; spec_idx < n_spectra ; spec_idx+=N_BLOCKS)
    {
        for(int freq_idx=threadIdx.x; freq_idx < spectrum_length ; freq_idx+=N_THREADS)
        {
            int idx=spec_idx*spectrum_length + freq_idx;
            //map unsigned shorts in to range [-0.5,0.5)
            df[idx] = ( (float)ds[idx] - 32768.0) / 65535.0;
        }
    }
}


__global__ void apply_weights(float *df, float *w, int n_spectra, int spectrum_length)
{
    for(int spec_idx=blockIdx.x; spec_idx < n_spectra ; spec_idx+=N_BLOCKS)
    {
        for(int freq_idx=threadIdx.x; freq_idx < spectrum_length ; freq_idx+=N_THREADS)
        {
            int idx=spec_idx*spectrum_length + freq_idx;
            df[idx] *= w[freq_idx];
        }
    }
}


/*
 transform and average signed data
*/

void process_vector_no_output(SAMPLE_TYPE *d_in, spectrometer_data *d)
{
    int n_spectra, data_length, spectrum_length;

    n_spectra=d->n_spectra;
    data_length=d->data_length;
    spectrum_length=d->spectrum_length;

    // ensure empty device spectrum
    if (hipMemsetAsync(d->d_spectrum, 0, sizeof(float)*(spectrum_length/2 + 1)) != hipSuccess)
    {
      fprintf(stderr, "Cuda error: Failed to zero device spectrum\n");
        exit(EXIT_FAILURE);
    }

    // ensure empty noise data acuumulation buffers
    if (hipMemsetAsync(d->d_out, 0, sizeof(float)*N_THREADS) != hipSuccess)
    {
      fprintf(stderr, "Cuda error: Failed to zero device spectrum\n");
        exit(EXIT_FAILURE);
    }

    // ensure empty noise data acuumulation buffers
    if (hipMemsetAsync(d->d_out2, 0, sizeof(float)*N_THREADS) != hipSuccess)
    {
      fprintf(stderr, "Cuda error: Failed to zero device spectrum\n");
        exit(EXIT_FAILURE);
    }

    // copy mem to device
    if (hipMemcpyAsync(d->ds_in, d_in, sizeof(SAMPLE_TYPE)*data_length, hipMemcpyHostToDevice) != hipSuccess)
    {
      fprintf(stderr, "Cuda error: Memory copy failed\n");
      exit(EXIT_FAILURE);
    }

    //convert datatype using GPU
    #ifdef HOSE_USE_ADQ7
    //convert signed ints to floats
    short_to_float_s<<< N_BLOCKS, N_THREADS >>>(d->ds_in, d->d_in, n_spectra, spectrum_length);
    #else
    //convert unsigned ints to floats
    short_to_float<<< N_BLOCKS, N_THREADS >>>(d->ds_in, d->d_in, n_spectra, spectrum_length);
    #endif

    //do the first pass parallel reduction of the data for the noise statistics
    cuda_noise_statistics_mbp_reduce1<<< N_BLOCKS, N_THREADS>>>(d->d_in, d->d_out, d->d_out2, data_length);

    //do the second pass parallel reduction of the data for the noise statistics
    cuda_noise_statistics_mbp_reduce2<<<1, N_THREADS>>>(d->d_out, d->d_out2, d->f_out, d->f_out2, N_THREADS);

    hipDeviceSynchronize();

    //copy the <x> and <x^2> values back to the host
    hipMemcpy(&d->sum, d->f_out, sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy(&d->sum2, d->f_out2, sizeof(float), hipMemcpyDeviceToHost );


    //have to apply the Blackman-Harris window function to the data now
    apply_weights<<< N_BLOCKS, N_THREADS >>>(d->d_in, d->d_window, n_spectra, spectrum_length);

    // cufft kernel execution
    if (hipfftExecR2C(d->plan, (float *)d->d_in, (hipfftComplex *)d->d_z_out)	!= HIPFFT_SUCCESS)
    {
        fprintf(stderr, "CUFFT error: ExecC2C Forward failed\n");
        exit(EXIT_FAILURE);
    }

    // this needs to be faster:
    square_and_accumulate_sum<<< 1, N_THREADS >>>(d->d_z_out, d->d_spectrum, n_spectra, spectrum_length/2+1);
    if (hipGetLastError() != hipSuccess)
    {
        fprintf(stderr, "Cuda error: Kernel failure, square_and_accumulate_sum\n");
        exit(EXIT_FAILURE);
    }

    // copying device resultant spectrum to host, now able to be manipulated
    if (hipMemcpy(d->spectrum, d->d_spectrum, sizeof(float) * spectrum_length/2, hipMemcpyDeviceToHost) != hipSuccess)
    {
        fprintf(stderr, "Cuda error: Memory copy failed, spectrum DtH\n");
        exit(EXIT_FAILURE);
    }

}
