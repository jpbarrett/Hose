#include "hip/hip_runtime.h"
#include "noise_statistics_mbp_reduce.h"

#include <cstdio>

static const int blockSize = 1024;
static const int gridSize = 24; //this number is hardware-dependent; usually #SM*2 is a good number.

//first pass reduction (compute partial sum and sum2)
__global__ void cuda_noise_statistics_mbp_reduce1(const float* in, float* out_sum, float* out_sum2, int n)
{
    int tid = threadIdx.x;
    int gid = tid + blockIdx.x*blockSize;
    const int gridSize = blockSize*gridDim.x;

    float sum = 0.0;
    float sum2 = 0.0;

    for (int i = gid; i < n; i += gridSize)
    {
        float tmp = in[i];
        sum += tmp;
        sum2 += tmp*tmp;
    }

    __shared__ float shared_workspace[blockSize];
    __shared__ float shared_workspace2[blockSize];

    shared_workspace[tid] = sum;
    shared_workspace2[tid] = sum2;

    //wait for all threads to sync up
    __syncthreads();

    //proceed in lock-step with parallel reduction
    for (int size = blockSize/2; size>0; size/=2)
    {
        if(tid<size)
        {
            shared_workspace[tid] += shared_workspace[tid+size];
            shared_workspace2[tid] += shared_workspace2[tid+size];
        }
        __syncthreads();
    }

    if(tid == 0)
    {
        out_sum[blockIdx.x] = shared_workspace[0];
        out_sum2[blockIdx.x] = shared_workspace2[0];
    }
};

__global__ void cuda_noise_statistics_mbp_reduce2(const float* in, const float* in2, float* out_sum, float* out_sum2, int n)
{
    int tid = threadIdx.x;

    __shared__ float shared_workspace[blockSize];
    __shared__ float shared_workspace2[blockSize];

    shared_workspace[tid] = in[tid];
    shared_workspace2[tid] = in2[tid];

    //wait for all threads to sync up
    __syncthreads();

    //proceed in lock-step with parallel reduction
    for (int size = blockSize/2; size>0; size/=2)
    {
        if(tid<size)
        {
            shared_workspace[tid] += shared_workspace[tid+size];
            shared_workspace2[tid] += shared_workspace2[tid+size];
        }
        __syncthreads();
    }

    if(tid == 0)
    {
        out_sum[0] = shared_workspace[0];
        out_sum2[0] = shared_workspace2[0];
    }
};



void noise_statistics_mbp_reduce(float* input, float* sum, float* sum2, int n)
{
    // Device input vectors
    float* d_in;
    float* d_out;
    float* d_out2;

    float* f_out;
    float* f_out2;

    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(float);

    // Allocate memory for each vector on GPU
    hipMalloc(&d_in, bytes);
    hipMalloc(&d_out, blockSize*sizeof(float));
    hipMalloc(&d_out2, blockSize*sizeof(float));
    hipMalloc(&f_out, sizeof(float));
    hipMalloc(&f_out2, sizeof(float));

    // Copy host input vector to device
    hipMemcpy(d_in, input, bytes, hipMemcpyHostToDevice);

    //run the first level of reduction
    cuda_noise_statistics_mbp_reduce1<<<gridSize, blockSize>>>(d_in, d_out, d_out2, n);

    //now reduce the partial results to a single number
    cuda_noise_statistics_mbp_reduce2<<<1, blockSize>>>(d_out, d_out2, f_out, f_out2, blockSize);
    //dev_out[0] now holds the final result
    hipDeviceSynchronize();

    // Copy array back to host --- just the first element
    hipMemcpy(sum, f_out, sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy(sum2, f_out2, sizeof(float), hipMemcpyDeviceToHost );
    //
    // float tmp = 0.0;
    // float tmp2 = 0.0;
    //
    // for(int i=0; i<blockSize; i++)
    // {
    //     tmp += sum[i];
    //     tmp2 += sum2[i];
    // }

    printf("sum %f \n",sum);
    printf("sum2 %f \n",sum2);

    // Release device memory
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out2);
    hipFree(f_out);
    hipFree(f_out2);
};
