#include "hip/hip_runtime.h"
#include <stdio.h>
#include "spectrometer.h"
#include <cmath>
#include <random>
#include <inttypes.h>
#include <chrono>
#include <iostream>
int main(int argc, char **argv)
{
  printf("running a benchmark\n");
  int veclen=2097152*64;
  int fftlen=2097152;
  spectrometer_data* sd = new_spectrometer_data(veclen, fftlen, 0);
  int16_t *noise; 
  std::random_device rd{};
  std::mt19937 gen{rd()};
  
  
  std::normal_distribution d{0.0,1.0};
  auto random_short = [&d, &gen]{return std::round(512*d(gen));};
  
  noise=(int16_t *)malloc(sizeof(int16_t)*veclen);
  
  for(int i=0; i<veclen; i++)
  {
	noise[i]=random_short();
  }
  FILE *o=fopen("noise.bin","wb");
  fwrite(noise,sizeof(int16_t),veclen,o);
  fclose(o);

  process_vector_no_output_(noise, sd, 0);
  o=fopen("spec.bin","wb");  
  fwrite(sd->spectrum,sizeof(float),sd->spectrum_length/2+1,o);
  fclose(o);
  
  typedef std::chrono::high_resolution_clock Time;
  typedef std::chrono::milliseconds ms;
  typedef std::chrono::duration<float> fsec;
  auto t0 = Time::now();
  
  

  
  long n_reps=10;
  long n_samples = n_reps*veclen;
  for(int i=0; i<n_reps; i++)
  {
    process_vector_no_output_(noise, sd, 0);
  }

  
  auto t1 = Time::now();
  fsec fs = t1 - t0;
  ms td = std::chrono::duration_cast<ms>(fs);
  double samps_per_sec=(double)n_samples/fs.count();
  std::cout << samps_per_sec << "samps per second\n";
  std::cout << fs.count() << "s\n";
  std::cout << td.count() << "ms\n";
}
