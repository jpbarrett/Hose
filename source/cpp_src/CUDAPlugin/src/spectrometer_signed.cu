#include "hip/hip_runtime.h"
/* Project that takes signal and calculates averaged power spectrum using NVIDIA
   CUDA GPGPU programming.

   Function main() tests empty data sets to check if the kernels
   (GPU-based algorithms), allocations, and data transfers are working properly.

   The central power of this program is from the kernels below tied together nicely
   in the process_vector() function.

   periodically wake up,
   scan the directory,
   read in data from file,
   and process...

   Juha Vierinen (x@mit.edu)
   Cory Cotter (optimization of square and accumulate sum)
*/
#include "spectrometer_signed.h"
#define MIN(X,Y) (((X) < (Y)) ? (X) : (Y))

/*
  create Blackmann-Harris window function
 */
void blackmann_harris_s(float* pOut, unsigned int num)
{
  const float a0      = 0.35875f;
  const float a1      = 0.48829f;
  const float a2      = 0.14128f;
  const float a3      = 0.01168f;

  unsigned int idx    = 0;
  while( idx < num )
  {
    pOut[idx]   = a0 - (a1 * cosf( (2.0f * M_PI * idx) / (num - 1) )) + (a2 * cosf( (4.0f * M_PI * idx) / (num - 1) )) - (a3 * cosf( (6.0f * M_PI * idx) / (num - 1) ));
    idx++;
  }
}

/*
  average spectra
 */
__global__ void square_and_accumulate_sum_s(const hipfftComplex* d_in,
					  float* d_out,
					  const unsigned n_spectra,
					  const unsigned spectrum_length)
{
    unsigned idx = threadIdx.x + blockDim.x*blockIdx.x;
    while (idx < spectrum_length)
    {
        float result = 0.0;
        for (int i = 0; i < n_spectra; i++)
        {
            int d_idx = i * spectrum_length + idx;
            result += d_in[d_idx].x * d_in[d_idx].x + d_in[d_idx].y * d_in[d_idx].y;
        }
        d_out[idx] = result;
        idx += gridDim.x * blockDim.x;
    }
}

/*
  convert int16_t data vector into single precision floating point
  also apply window function *w
 */
__global__ void short_to_float_s(int16_t *ds, float *df, float *w, int n_spectra, int spectrum_length)
{
  for(int spec_idx=blockIdx.x; spec_idx < n_spectra ; spec_idx+=N_BLOCKS_S)
  {
    for(int freq_idx=threadIdx.x; freq_idx < spectrum_length ; freq_idx+=N_THREADS_S)
    {
      int idx=spec_idx*spectrum_length + freq_idx;
      df[idx] = w[freq_idx]*((float)ds[idx])/65532.0;
    }
  }
}

spectrometer_output_s *new_spectrometer_output_s()
{
  spectrometer_output_s *o;
  o=(spectrometer_output_s *)malloc(sizeof(spectrometer_output_s));
  o->out=NULL;
  o->ut_sec=0;
  o->year=0;
  o->month=0;
  o->day=0;
  return(o);
}

int cmpfunc_s(const void *a, const void *b)
{
  if( (*(int64_t *)a - *(int64_t *)b) < 0)
    return(-1);
  if( (*(int64_t *)a - *(int64_t *)b) > 0)
    return(1);
  else
    return(0);
}

/*
   figure out how many files in /ram

 */
void free_files_s(spectrometer_files_s *f)
{
  free(f->file_numbers);
  free(f);
}

spectrometer_files_s *get_files_s(char const *dname)
{
  DIR *d;
  spectrometer_files_s *f;
  struct dirent *dir;
  struct stat st;
  char fname[4096];
  f=(spectrometer_files_s *)malloc(sizeof(spectrometer_files_s));
  int size;
  int fsize;
  int n_files;
  uint64_t number;
  n_files=0;
  // out-586888628755720000.bin
  fsize=0;
  d=opendir(dname);
  if(d)
  {
    while((dir=readdir(d)) != NULL)
    {
      if(strlen(dir->d_name) == 26 && dir->d_name[0]=='o')
      {
	sprintf(fname,"%s/%s",dname,dir->d_name);
	stat(fname,&st);
	size=st.st_size;
	if(size > fsize)
	{
	  fsize=size;
	}
	n_files++;
      }
    }
    closedir(d);
  }

  f->n_files=n_files;
  f->file_size=fsize;
  // there might be some new files, pad for extra room
  f->file_numbers=(uint64_t *)malloc(sizeof(uint64_t)*(n_files+1000));
  d=opendir(dname);
  n_files=0;
  if(d)
  {
    while((dir=readdir(d)) != NULL)
    {
      if(strlen(dir->d_name) == 26 && dir->d_name[0]=='o' && n_files < f->n_files)
      {
	sprintf(fname,"%s/%s",dname,dir->d_name);
	stat(fname,&st);
	size=st.st_size;
	if(size==fsize)
	{
	  sscanf(dir->d_name,"out-%"PRIu64".bin",&number);
	  f->file_numbers[n_files]=number;
	  n_files++;
	}
      }
    }
    closedir(d);
  }
  f->n_files=n_files;
  if(f->n_files > 1)
  {
    qsort(f->file_numbers, f->n_files, sizeof(uint64_t), cmpfunc_s);
  }

  return(f);
}
/*
      {
	printf("reading data %d\n",size/2);
	data_length=size/2;
	n_spectra=data_length/spectrum_length;
	r_in = (int16_t *)malloc(data_length*sizeof(int16_t));
	in=(FILE *)fopen(argv[1],"r");
	fread(r_in,sizeof(int16_t),data_length,in);
	fclose(in);
      }
      else
	exit(0);
    }else{
*/

// void day_dirname_s(spectrometer_output_s *o, char const *prefix)
// {
//   struct tm dt;
//   time_t t;
//   char cmd[4096];
//   int year, month, day;
// 
//   time(&t);
//   gmtime_r(&t, &dt);
//   year=dt.tm_year+1900;
//   month=dt.tm_mon+1;
//   day=dt.tm_mday;
//   sprintf(o->dname,"%s/%d.%02d.%02d",prefix, year, month, day);
//   if(o->year != year || o->day != day || o->month != month)
//   {
//     // turns out this operation is _super_ slow. avoid doing it too often.
//     sprintf(cmd,"mkdir -p %s",o->dname);
//     printf("creating directory %s\n",o->dname);
//     system(cmd);
//   }
//   o->year=year;
//   o->month=month;
//   o->day=day;
// }

void gen_test_signal_s(int16_t *r_in, int spectrum_length, int n_spectra)
{
  // test signal
  for(int ti=0; ti<spectrum_length; ti++)
    r_in[ti]=(int16_t) (sinf(2.0*M_PI*10.0*(float)ti/((float)spectrum_length))*256.0 + 32768.0 );

  for(int i=1; i<n_spectra; i++)
    for(int ti=0; ti<spectrum_length; ti++)
      r_in[i*spectrum_length + ti]=r_in[ti];

#ifdef WRITE_INPUT
  FILE *out;
  out=fopen("in.bin","w");
  fwrite(r_in,sizeof(int16_t),10.0*spectrum_length,out);
  fclose(out);
#endif

}

/*
  TODO: this could be double buffered to increase speed:
  - read file in background, while processing the previous one
  - could be done with multithreading
 */

extern "C" spectrometer_data_s *new_spectrometer_data_s(int data_length, int spectrum_length)
{
  spectrometer_data_s *d;
  int n_spectra;
  n_spectra=data_length/spectrum_length;

  unsigned long available, total;
  hipMemGetInfo(&available, &total);
    printf("gpu mem: avail and total = %lu and %lu \n", available, total);


  d=(spectrometer_data_s *)malloc(sizeof(spectrometer_data_s));

  // result on the cpu
  d->spectrum = (float *)malloc( (spectrum_length/2+1) * sizeof(float));

  d->data_length=data_length;
  d->spectrum_length=spectrum_length;
  d->n_spectra=n_spectra;

  d->window = (float *)malloc(spectrum_length*sizeof(float));
  blackmann_harris_s(d->window,spectrum_length);

  // allocating device memory to the above pointers
  // reserve extra for +1 in place transforms
    unsigned long want =  sizeof(hipfftComplex)*n_spectra*(spectrum_length/2 + 1);
    printf("required gpu buff size = %lu \n", want);


  int code = hipMalloc( (void **) &d->d_in, sizeof(hipfftComplex)*n_spectra*(spectrum_length/2 + 1) );
  if( code != hipSuccess)
  {
    printf("code = %d\n", code);
    fprintf(stderr, "Cuda error: Failed to allocate input data vector\n");
    exit(EXIT_FAILURE);
  }

  hipMemGetInfo(&available, &total);
    printf("gpu mem: avail and total = %lu and %lu\n", available, total);


  code = hipMalloc((void **) &d->d_z_out, sizeof(hipfftComplex)*n_spectra*(spectrum_length/2 + 1));
  if ( code != hipSuccess)
  {
    printf("code = %d\n", code);
    fprintf(stderr, "Cuda error: Failed to allocate input data vector\n");
    exit(EXIT_FAILURE);
  }

  hipMemGetInfo(&available, &total);
    printf("gpu mem: avail and total = %lu and %lu\n", available, total);


    code = hipMalloc((void **) &d->d_window, sizeof(float)*spectrum_length);
  if (code != hipSuccess)
  {
    printf("code = %d\n", code);
    fprintf(stderr, "Cuda error: Failed to allocate input data vector\n");
    exit(EXIT_FAILURE);
  }

  hipMemGetInfo(&available, &total);
    printf("gpu mem: avail and total = %lu and %lu\n", available, total);


  // in-place seems to have a bug that causes the end to be garbled.
  //  d->d_z_out =(hipfftComplex *) d->d_in;

    code = hipMalloc((void **) &d->ds_in,sizeof(int16_t)*data_length);
  if (code != hipSuccess)
  {
    printf("code = %d\n", code);
    fprintf(stderr, "Cuda error: Failed to allocate input data vector\n");
    exit(EXIT_FAILURE);
  }

  hipMemGetInfo(&available, &total);
    printf("gpu mem: avail and total = %lu and %lu\n", available, total);


  if (hipMalloc((void **) &d->d_spectrum,sizeof(float)*(spectrum_length/2+1))
      != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate spectrum\n");
    exit(EXIT_FAILURE);
  }

  hipMemGetInfo(&available, &total);
    printf("gpu mem: avail and total = %lu and %lu\n", available, total);


  // initializing 1D FFT plan, this will tell cufft execution how to operate
  // cufft is well optimized and will run with different parameters than above
  //    hipfftHandle plan;
    
  if (hipfftPlan1d(&d->plan, spectrum_length, HIPFFT_R2C, n_spectra) != HIPFFT_SUCCESS)
  {
    fprintf(stderr, "CUFFT error: Plan creation failed\n");
    fprintf(stderr, "spec len, n spec = %d, %d", spectrum_length, n_spectra);
    exit(EXIT_FAILURE);
  }

  // copy window to device
  if (hipMemcpy(d->d_window, d->window, sizeof(float)*spectrum_length, hipMemcpyHostToDevice) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Memory copy failed, window function HtD\n");
    exit(EXIT_FAILURE);
  }
  return(d);
}

extern "C" void free_spectrometer_data_s(spectrometer_data_s *d)
{
  if (hipFree(d->d_in) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free input\n");
    exit(EXIT_FAILURE);
  }
  free(d->window);
  free(d->spectrum);
  if (hipFree(d->d_window) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free window function\n");
    exit(EXIT_FAILURE);
  }
  if (hipFree(d->d_z_out) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free input\n");
    exit(EXIT_FAILURE);
  }
  if (hipFree(d->ds_in) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free input\n");
    exit(EXIT_FAILURE);
  }

  if (hipFree(d->d_spectrum) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free spec\n");
    exit(EXIT_FAILURE);
  }
  if (hipfftDestroy(d->plan) != HIPFFT_SUCCESS) {
    fprintf(stderr, "CUFFT error: Failed to destroy plan\n");
    exit(EXIT_FAILURE);
  }
  free(d);
}

extern "C" void process_vector_no_output_s(int16_t *d_in, spectrometer_data_s *d, uint64_t t0)
{
    int n_spectra, data_length, spectrum_length;
    uint64_t ut_sec;

    n_spectra=d->n_spectra;
    data_length=d->data_length;
    spectrum_length=d->spectrum_length;


    // ensure empty device spectrum
    if (hipMemsetAsync(d->d_spectrum, 0, sizeof(float)*(spectrum_length/2 + 1)) != hipSuccess)
    {
      fprintf(stderr, "Cuda error: Failed to zero device spectrum\n");
        exit(EXIT_FAILURE);
    }

    // copy mem to device
    if (hipMemcpyAsync(d->ds_in, d_in, sizeof(int16_t)*data_length, hipMemcpyHostToDevice) != hipSuccess)
    {
      fprintf(stderr, "Cuda error: Memory copy failed, tx HtD\n");
      exit(EXIT_FAILURE);
    }

    // convert datatype using GPU
    short_to_float_s<<< N_BLOCKS_S, N_THREADS_S >>>(d->ds_in, d->d_in, d->d_window, n_spectra, spectrum_length);

    // cufft kernel execution
    if (hipfftExecR2C(d->plan, (float *)d->d_in, (hipfftComplex *)d->d_z_out)
	!= HIPFFT_SUCCESS)
    {
      fprintf(stderr, "CUFFT error: ExecC2C Forward failed\n");
      exit(EXIT_FAILURE);
    }

    // this needs to be faster:
    square_and_accumulate_sum_s<<< 1, N_THREADS_S >>>(d->d_z_out, d->d_spectrum, n_spectra, spectrum_length/2+1);
    if (hipGetLastError() != hipSuccess) {
       fprintf(stderr, "Cuda error: Kernel failure, square_and_accumulate_sum\n");
       exit(EXIT_FAILURE);
    }

    // copying device resultant spectrum to host, now able to be manipulated
    if (hipMemcpy(d->spectrum, d->d_spectrum, sizeof(float) * spectrum_length/2,
        hipMemcpyDeviceToHost) != hipSuccess)
    {
        fprintf(stderr, "Cuda error: Memory copy failed, spectrum DtH\n");
        exit(EXIT_FAILURE);
    }

}

void cuda_alloc_pinned_memory( void** ptr, size_t s)
{
    int code = hipHostMalloc(ptr, s);
    if(code != hipSuccess)
    {
        //set pointer to null if there is an error
        fprintf(stderr, "Cuda error: hipHostMalloc failed.\n");
        *ptr = NULL;
    }
}

void cuda_free_pinned_memory(void* ptr)
{
    int code = hipHostFree(ptr);
    if(code != hipSuccess)
    {
        fprintf(stderr, "Cuda error: hipHostFree failed.\n");
    }
}
