#include "hip/hip_runtime.h"
/* Project that takes signal and calculates averaged power spectrum using NVIDIA
   CUDA GPGPU programming. 

   Function main() tests empty data sets to check if the kernels 
   (GPU-based algorithms), allocations, and data transfers are working properly.

   The central power of this program is from the kernels below tied together nicely
   in the process_vector() function.

   periodically wake up, 
   scan the directory, 
   read in data from file, 
   and process...

   Juha Vierinen (x@mit.edu)
   Cory Cotter (optimization of square and accumulate sum)
*/
#include "spectrometer.h"
#define MIN(X,Y) (((X) < (Y)) ? (X) : (Y))

/*
  create Blackmann-Harris window function
 */
void blackmann_harris(float* pOut, unsigned int num)
{
  const float a0      = 0.35875f;
  const float a1      = 0.48829f;
  const float a2      = 0.14128f;
  const float a3      = 0.01168f;

  unsigned int idx    = 0;
  while( idx < num )
  {
    pOut[idx]   = a0 - (a1 * cosf( (2.0f * M_PI * idx) / (num - 1) )) + (a2 * cosf( (4.0f * M_PI * idx) / (num - 1) )) - (a3 * cosf( (6.0f * M_PI * idx) / (num - 1) ));
    idx++;
  }
}

/*
  average spectra
 */
__global__ void square_and_accumulate_sum(const hipfftComplex* d_in, 
					  float* d_out, 
					  const unsigned n_spectra, 
					  const unsigned spectrum_length)
{
    unsigned idx = threadIdx.x + blockDim.x*blockIdx.x; 
    while (idx < spectrum_length) 
    { 
        float result = 0.0; 
        for (int i = 0; i < n_spectra; i++) 
        { 
            int d_idx = i * spectrum_length + idx; 
            result += d_in[d_idx].x * d_in[d_idx].x + d_in[d_idx].y * d_in[d_idx].y; 
        }   
        d_out[idx] = result; 
        idx += gridDim.x * blockDim.x; 
    }   
}

/*
  convert uint16_t data vector into single precision floating point
  also apply window function *w
 */
__global__ void short_to_float(uint16_t *ds, float *df, float *w, int n_spectra, int spectrum_length)
{
  for(int spec_idx=blockIdx.x; spec_idx < n_spectra ; spec_idx+=N_BLOCKS)
  {
    for(int freq_idx=threadIdx.x; freq_idx < spectrum_length ; freq_idx+=N_THREADS)
    {
      int idx=spec_idx*spectrum_length + freq_idx;
      df[idx] = w[freq_idx]*((float)ds[idx]-32768.0)/65532.0;
    }
  }
}

spectrometer_output *new_spectrometer_output()
{
  spectrometer_output *o;
  o=(spectrometer_output *)malloc(sizeof(spectrometer_output));
  o->out=NULL;
  o->ut_sec=0;
  o->year=0;
  o->month=0;
  o->day=0;
  return(o);
}

int cmpfunc(const void *a, const void *b)
{
  if( (*(int64_t *)a - *(int64_t *)b) < 0)
    return(-1);
  if( (*(int64_t *)a - *(int64_t *)b) > 0)
    return(1);
  else
    return(0);
}

/* 
   figure out how many files in /ram
   
 */
void free_files(spectrometer_files *f)
{
  free(f->file_numbers);
  free(f);
}

spectrometer_files *get_files(char const *dname)
{
  DIR *d;
  spectrometer_files *f;
  struct dirent *dir;
  struct stat st;
  char fname[4096];
  f=(spectrometer_files *)malloc(sizeof(spectrometer_files));
  int size;
  int fsize;
  int n_files;
  uint64_t number;
  n_files=0;
  // out-586888628755720000.bin
  fsize=0;
  d=opendir(dname);
  if(d)
  {
    while((dir=readdir(d)) != NULL)
    {
      if(strlen(dir->d_name) == 26 && dir->d_name[0]=='o')
      {
	sprintf(fname,"%s/%s",dname,dir->d_name);
	stat(fname,&st);
	size=st.st_size;
	if(size > fsize)
	{
	  fsize=size;
	}
	n_files++;
      }
    }
    closedir(d);
  }

  f->n_files=n_files;
  f->file_size=fsize;
  // there might be some new files, pad for extra room
  f->file_numbers=(uint64_t *)malloc(sizeof(uint64_t)*(n_files+1000));
  d=opendir(dname);
  n_files=0;
  if(d)
  {
    while((dir=readdir(d)) != NULL)
    {
      if(strlen(dir->d_name) == 26 && dir->d_name[0]=='o' && n_files < f->n_files)
      {
	sprintf(fname,"%s/%s",dname,dir->d_name);
	stat(fname,&st);
	size=st.st_size;
	if(size==fsize)
	{
	  sscanf(dir->d_name,"out-%"PRIu64".bin",&number);
	  f->file_numbers[n_files]=number;
	  n_files++;
	}
      }
    }
    closedir(d);
  }
  f->n_files=n_files;
  if(f->n_files > 1)
  {
    qsort(f->file_numbers, f->n_files, sizeof(uint64_t), cmpfunc);
  }

  return(f);
}
/*
      {
	printf("reading data %d\n",size/2);
	data_length=size/2;
	n_spectra=data_length/spectrum_length;
	r_in = (uint16_t *)malloc(data_length*sizeof(uint16_t));
	in=(FILE *)fopen(argv[1],"r");
	fread(r_in,sizeof(uint16_t),data_length,in);
	fclose(in);
      }
      else
	exit(0);
    }else{
*/

void day_dirname(spectrometer_output *o, char const *prefix)
{
  struct tm dt;
  time_t t;
  char cmd[4096];
  int year, month, day;

  time(&t);
  gmtime_r(&t, &dt);
  year=dt.tm_year+1900;
  month=dt.tm_mon+1;
  day=dt.tm_mday;
  sprintf(o->dname,"%s/%d.%02d.%02d",prefix, year, month, day);
  if(o->year != year || o->day != day || o->month != month)
  {
    // turns out this operation is _super_ slow. avoid doing it too often.
    sprintf(cmd,"mkdir -p %s",o->dname);
    printf("creating directory %s\n",o->dname);
    system(cmd);
  }
  o->year=year;
  o->month=month;
  o->day=day;
}

void gen_test_signal(uint16_t *r_in, int spectrum_length, int n_spectra)
{
  // test signal
  for(int ti=0; ti<spectrum_length; ti++)
    r_in[ti]=(uint16_t) (sinf(2.0*M_PI*10.0*(float)ti/((float)spectrum_length))*256.0 + 32768.0 );
  
  for(int i=1; i<n_spectra; i++)
    for(int ti=0; ti<spectrum_length; ti++) 
      r_in[i*spectrum_length + ti]=r_in[ti];

#ifdef WRITE_INPUT 
  FILE *out;
  out=fopen("in.bin","w");
  fwrite(r_in,sizeof(uint16_t),10.0*spectrum_length,out);
  fclose(out);
#endif

}

/*
  TODO: this could be double buffered to increase speed:
  - read file in background, while processing the previous one
  - could be done with multithreading
 */
int process(char const *dname, char const *dname_results)
{
    // 32k transform
    int spectrum_length = SPECTRUM_LENGTH;
    //int n_spectra;//=2440; // Limited by the amount of RAM on the GPU.
    int data_length;// = n_spectra*spectrum_length;
    spectrometer_files *f;
    char fname[4096];
    spectrometer_data *d;
    spectrometer_output *o;
    uint16_t *r_in;
    FILE *in;

    f=get_files(dname);
    if(f->n_files == 0 || f->file_size==0)
    {
      printf("waiting for more data...\n");
      sleep(1);
      return(0);
    }

    data_length=f->file_size/2-BLANKING;
    //n_spectra=data_length/spectrum_length;

    // real valued input signal as short ints
    r_in = (uint16_t *)malloc(data_length*sizeof(uint16_t));

#ifdef TEST_SIGNAL
    gen_test_signal(r_in, spectrum_length, n_spectra);
#endif

    d=new_spectrometer_data(data_length,spectrum_length);
    o=new_spectrometer_output();
    
    for(int i=0; i < f->n_files; i++)
    {
#ifdef TIMING_PRINT
      clock_t start, end;
      clock_t r_end;
      start=clock();
#endif
      sprintf(fname,"%s/out-%"PRIu64".bin",dname, f->file_numbers[i]);
      
      if(in=fopen(fname,"r"))
      {
	// blanking, 2 bytes per sample
	fseek(in,BLANKING*2,SEEK_SET);
	size_t rsize=fread(r_in,sizeof(uint16_t),data_length,in);
#ifdef TIMING_PRINT
	r_end=clock();
#endif
	if(rsize == data_length)
	  process_vector(r_in, d, f->file_numbers[i], o, dname_results);	
	else
	  printf("couldn't read file\n");

	fclose(in);
      }
      else
	printf("couldn't open file %s\n", fname);

      // delete file
      sprintf(fname,"rm %s/out-%"PRIu64".bin", dname, f->file_numbers[i]);
      system(fname);
    
#ifdef TIMING_PRINT
      end=clock();
      double dt = ((double) (end-start))/CLOCKS_PER_SEC;
      double r_dt = ((double) (r_end-start))/CLOCKS_PER_SEC;
      
      if(i > 1){
	printf("Time elapsed %1.3f/%1.3f s read %1.3f%% speed ratio %1.3f t %"PRIu64" dt %"PRIu64"\n", dt, (double)(data_length+BLANKING)/SAMPLE_RATE,
	       (r_dt/dt)*100.0,
	       ((double)(data_length+BLANKING)/SAMPLE_RATE)  / dt / 2, //halve the speed ratio for two pol 
	       f->file_numbers[i],f->file_numbers[i]-f->file_numbers[i-1]);
      }
#endif
    }

    free_files(f);
    free_spectrometer_data(d);
    free(r_in);
    return 0;
}

extern "C" spectrometer_data *new_spectrometer_data(int data_length, int spectrum_length)
{
  spectrometer_data *d;
  int n_spectra;
  n_spectra=data_length/spectrum_length;

  d=(spectrometer_data *)malloc(sizeof(spectrometer_data));

  // result on the cpu
  d->spectrum = (float *)malloc( (spectrum_length/2+1) * sizeof(float));

  d->data_length=data_length;
  d->spectrum_length=spectrum_length;
  d->n_spectra=n_spectra;

  d->window = (float *)malloc(spectrum_length*sizeof(float));
  blackmann_harris(d->window,spectrum_length);

  // allocating device memory to the above pointers
  // reserve extra for +1 in place transforms
  if (hipMalloc((void **) &d->d_in, sizeof(hipfftComplex)*n_spectra*(spectrum_length/2 + 1)) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate input data vector\n");
    exit(EXIT_FAILURE);
  }
  if (hipMalloc((void **) &d->d_z_out, sizeof(hipfftComplex)*n_spectra*(spectrum_length/2 + 1)) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate input data vector\n");
    exit(EXIT_FAILURE);
  }

  if (hipMalloc((void **) &d->d_window, sizeof(float)*spectrum_length) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate input data vector\n");
    exit(EXIT_FAILURE);
  }

  // in-place seems to have a bug that causes the end to be garbled.
  //  d->d_z_out =(hipfftComplex *) d->d_in;

  if (hipMalloc((void **) &d->ds_in,sizeof(uint16_t)*data_length) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate input data vector\n");
    exit(EXIT_FAILURE);
  }

  if (hipMalloc((void **) &d->d_spectrum,sizeof(float)*(spectrum_length/2+1))
      != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate spectrum\n");
    exit(EXIT_FAILURE);
  }

  // initializing 1D FFT plan, this will tell cufft execution how to operate
  // cufft is well optimized and will run with different parameters than above
  //    hipfftHandle plan;
  if (hipfftPlan1d(&d->plan, spectrum_length, HIPFFT_R2C, n_spectra) != HIPFFT_SUCCESS) 
  {
    fprintf(stderr, "CUFFT error: Plan creation failed\n");
    exit(EXIT_FAILURE);
  }

  // copy window to device
  if (hipMemcpy(d->d_window, d->window, sizeof(float)*spectrum_length, hipMemcpyHostToDevice) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Memory copy failed, window function HtD\n");
    exit(EXIT_FAILURE);
  }
  return(d);
}

extern "C" void free_spectrometer_data(spectrometer_data *d)
{
  if (hipFree(d->d_in) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free input\n");
    exit(EXIT_FAILURE);
  }
  free(d->window);
  free(d->spectrum);
  if (hipFree(d->d_window) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free window function\n");
    exit(EXIT_FAILURE);
  }
  if (hipFree(d->d_z_out) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free input\n");
    exit(EXIT_FAILURE);
  }
  if (hipFree(d->ds_in) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free input\n");
    exit(EXIT_FAILURE);
  }

  if (hipFree(d->d_spectrum) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free spec\n");
    exit(EXIT_FAILURE);
  }
  if (hipfftDestroy(d->plan) != HIPFFT_SUCCESS) {
    fprintf(stderr, "CUFFT error: Failed to destroy plan\n");
    exit(EXIT_FAILURE);
  }
  free(d);
}

/* This is the primary function in this program, meant to be embedded into other
**  programs. The transmit signal, tx, should be complex conjugated prior to use here.
**  The float types for tx and echo are useful when this function is embedded; the extern
**  "C" is also here for that purpose. Process_echoes sets up and runs the kernels on
**  the GPU, complex_mult, a 1D FFT, and a spectrum accumulation. Host spectrum is not
**  freed, so as to be taken and analyzed.
*/
//    process_vector((float *)z_in, data_length, spectrum, spectrum_length);



extern "C" void process_vector(uint16_t *d_in, spectrometer_data *d, uint64_t t0, spectrometer_output *o, char const *dname_results)
{
    int n_spectra, data_length, spectrum_length;
    //    FILE *out;
    uint64_t ut_sec;
    //    char fname[4096];
    //char dname[4096];

    n_spectra=d->n_spectra;
    data_length=d->data_length;
    spectrum_length=d->spectrum_length;

#ifdef DEBUG_Z_OUT
    // debug out
    hipfftComplex *z_out = (hipfftComplex *)malloc( n_spectra*(spectrum_length/2 + 1)*sizeof(hipfftComplex));
#endif
    // ensure empty device spectrum
    if (hipMemset(d->d_spectrum, 0, sizeof(float)*(spectrum_length/2 + 1)) != hipSuccess)
    {
      fprintf(stderr, "Cuda error: Failed to zero device spectrum\n");
        exit(EXIT_FAILURE);
    }

    // copy mem to device
    if (hipMemcpy(d->ds_in, d_in, sizeof(uint16_t)*data_length, hipMemcpyHostToDevice) != hipSuccess)
    {
      fprintf(stderr, "Cuda error: Memory copy failed, tx HtD\n");
      exit(EXIT_FAILURE);
    }
    
    // convert datatype using GPU
    short_to_float<<< N_BLOCKS, N_THREADS >>>(d->ds_in, d->d_in, d->d_window, n_spectra, spectrum_length);

    // cufft kernel execution
    if (hipfftExecR2C(d->plan, (float *)d->d_in, (hipfftComplex *)d->d_z_out)
	!= HIPFFT_SUCCESS)
    {
      fprintf(stderr, "CUFFT error: ExecC2C Forward failed\n");
      exit(EXIT_FAILURE);
    }

    // copying device resultant spectrum to host, now able to be manipulated
    // debug 
#ifdef DEBUG_Z_OUT
    hipMemcpy(z_out, d->d_z_out, sizeof(hipfftComplex)*n_spectra*(spectrum_length/2 + 1 ), hipMemcpyDeviceToHost);
    out=fopen("z_out.bin","w");
    fwrite(z_out,sizeof(hipfftComplex),n_spectra*(spectrum_length/2 + 1),out);
    fclose(out);
#endif

    // this needs to be faster:
    square_and_accumulate_sum<<< 1, N_THREADS >>>(d->d_z_out, d->d_spectrum, n_spectra, spectrum_length/2+1);
    if (hipGetLastError() != hipSuccess) {
       fprintf(stderr, "Cuda error: Kernel failure, square_and_accumulate_sum\n");
       exit(EXIT_FAILURE);
    }

    // copying device resultant spectrum to host, now able to be manipulated
    if (hipMemcpy(d->spectrum, d->d_spectrum, sizeof(float) * spectrum_length/2,
        hipMemcpyDeviceToHost) != hipSuccess)
    {
        fprintf(stderr, "Cuda error: Memory copy failed, spectrum DtH\n");
        exit(EXIT_FAILURE);
    }

    ut_sec=t0/SAMPLE_RATE;
    if(o->ut_sec != ut_sec || o->out == NULL)
    {
      // only make directory when necessary
      day_dirname(o, dname_results);
      // write results into one file per second
      sprintf(o->fname,"%s/spec-%"PRIu64".bin",o->dname, ut_sec);
      o->out=fopen(o->fname,"a");
      o->ut_sec=ut_sec;
    }
    fwrite(d->spectrum,sizeof(float),spectrum_length/2+1,o->out);
    fclose(o->out);
    o->out=NULL;

}
